#include "hip/hip_runtime.h"
/* 
 * SPDX-FileCopyrightText: Copyright (c) 2017,2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <sstream>

#include <mpi.h>

#define MPI_CALL(call)                                                                \
    {                                                                                 \
        int mpi_status = call;                                                        \
        if (0 != mpi_status) {                                                        \
            char mpi_error_string[MPI_MAX_ERROR_STRING];                              \
            int mpi_error_string_length = 0;                                          \
            MPI_Error_string(mpi_status, mpi_error_string, &mpi_error_string_length); \
            if (NULL != mpi_error_string)                                             \
                fprintf(stderr,                                                       \
                        "ERROR: MPI call \"%s\" in line %d of file %s failed "        \
                        "with %s "                                                    \
                        "(%d).\n",                                                    \
                        #call, __LINE__, __FILE__, mpi_error_string, mpi_status);     \
            else                                                                      \
                fprintf(stderr,                                                       \
                        "ERROR: MPI call \"%s\" in line %d of file %s failed "        \
                        "with %d.\n",                                                 \
                        #call, __LINE__, __FILE__, mpi_status);                       \
        }                                                                             \
    }

#include <hip/hip_runtime.h>

//TODO: Include NVSHMEM headers
#ifdef SOLUTION
#include <nvshmem.h>
#include <nvshmemx.h>
#endif

#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

#ifdef USE_NVTX
#include <nvToolsExt.h>

const uint32_t colors[] = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff,
                           0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color_id = cid;                                \
        color_id = color_id % num_colors;                  \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }

#ifdef USE_DOUBLE
typedef double real;
#define MPI_REAL_TYPE MPI_DOUBLE
#else
typedef float real;
#define MPI_REAL_TYPE MPI_FLOAT
#endif

constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
#ifdef SOLUTION
                              const int iy_end, const int nx, const bool calculate_norm,
                              const int top, const int iy_top_lower_boundary_idx,
                              const int bottom, const int iy_bottom_upper_boundary_idx) {
#else
                              const int iy_end, const int nx, const bool calculate_norm) {
#endif
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;
        if (calculate_norm) {
            real residue = new_val - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }

        //TODO: push values near boundary to top and bottom PE, remember to change the signature of
        //      jacobi_kernel
#ifdef SOLUTION
        if (iy_start == iy) {
            nvshmem_float_p(a_new + iy_top_lower_boundary_idx * nx + ix, new_val, top);
        }
        if ((iy_end - 1) == iy) {
            nvshmem_float_p(a_new + iy_bottom_upper_boundary_idx * nx + ix, new_val, bottom);
        }
#endif
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print);

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

int main(int argc, char* argv[]) {
    MPI_CALL(MPI_Init(&argc, &argv));
    int rank;
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    int size;
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));
    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    int local_rank = -1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }

    CUDA_RT_CALL(hipSetDevice(local_rank%num_devices));
    CUDA_RT_CALL(hipFree(0));

    //TODO: Initialize NVSHMEM using nvshmemx_init_attr
#ifdef SOLUTION
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;
    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    assert( size == nvshmem_n_pes() );
    assert( rank == nvshmem_my_pe() );
#endif

    real* a_ref_h;
    CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
    real* a_h;
    CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));
    double runtime_serial = single_gpu(nx, ny, iter_max, a_ref_h, nccheck, !csv && (0 == rank));

    // ny - 2 rows are distributed amongst `size` ranks in such a way
    // that each rank gets either (ny - 2) / size or (ny - 2) / size + 1 rows.
    // This optimizes load balancing when (ny - 2) % size != 0
    int chunk_size;
    int chunk_size_low = (ny - 2) / size;
    int chunk_size_high = chunk_size_low + 1;
    // To calculate the number of ranks that need to compute an extra row,
    // the following formula is derived from this equation:
    // num_ranks_low * chunk_size_low + (size - num_ranks_low) * (chunk_size_low + 1) = ny - 2
    int num_ranks_low = size * chunk_size_low + size -
                        (ny - 2);  // Number of ranks with chunk_size = chunk_size_low
    if (rank < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    //TODO: Allocate a and a_new from the NVSHMEM symmetric heap
    //      Note: size needs to be the same on all PEs but chunk_size might not be!
#ifdef SOLUTION
    real* a = (real*) nvshmem_malloc(nx * (chunk_size_high + 2) * sizeof(real));
    real* a_new = (real*) nvshmem_malloc(nx * (chunk_size_high + 2) * sizeof(real));
#else
    real* a;
    CUDA_RT_CALL(hipMalloc(&a, nx * (chunk_size + 2) * sizeof(real)));
    real* a_new;
    CUDA_RT_CALL(hipMalloc(&a_new, nx * (chunk_size + 2) * sizeof(real)));
#endif    

    CUDA_RT_CALL(hipMemset(a, 0, nx * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * (chunk_size + 2) * sizeof(real)));

    // Calculate local domain boundaries
    int iy_start_global;  // My start index in the global array
    if (rank < num_ranks_low) {
        iy_start_global = rank * chunk_size_low + 1;
    } else {
        iy_start_global =
            num_ranks_low * chunk_size_low + (rank - num_ranks_low) * chunk_size_high + 1;
    }
    int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

    int iy_start = 1;
    int iy_end = iy_start + chunk_size;

    const int top = rank > 0 ? rank - 1 : (size - 1);
    const int bottom = (rank + 1) % size;

    //TODO: calculate halo/boundary row index of top and bottom neighbors
#ifdef SOLUTION
    const int iy_top_lower_boundary_idx = (top < num_ranks_low) ? (chunk_size_low + 1) : (chunk_size_high + 1);
    const int iy_bottom_upper_boundary_idx = 0;
#endif

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<(chunk_size + 2) / 128 + 1, 128>>>(a, a_new, PI, iy_start_global - 1, nx, (chunk_size + 2), ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    hipStream_t compute_stream;
    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    hipEvent_t compute_done;
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));

    real* l2_norm_d;
    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    real* l2_norm_h;
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    PUSH_RANGE("MPI_Warmup", 5)
    for (int i = 0; i < 10; ++i) {
        const int top = rank > 0 ? rank - 1 : (size - 1);
        const int bottom = (rank + 1) % size;
        MPI_CALL(MPI_Sendrecv(a_new + iy_start * nx, nx, MPI_REAL_TYPE, top, 0,
                              a_new + (iy_end * nx), nx, MPI_REAL_TYPE, bottom, 0, MPI_COMM_WORLD,
                              MPI_STATUS_IGNORE));
        MPI_CALL(MPI_Sendrecv(a_new + (iy_end - 1) * nx, nx, MPI_REAL_TYPE, bottom, 0, a_new, nx,
                              MPI_REAL_TYPE, top, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));
        std::swap(a_new, a);
    }
    POP_RANGE

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (!csv && 0 == rank) {
        printf(
            "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
            "every %d iterations\n",
            iter_max, ny, nx, nccheck);
    }

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                  ((iy_end - iy_start) + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    real l2_norm = 1.0;
    bool calculate_norm;  // boolean to store whether l2 norm will be calculated in
                          //   an iteration or not

    MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
    double start = MPI_Wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        calculate_norm = (iter % nccheck) == 0 || (!csv && (iter % 100) == 0);

        //TODO: pass top and bottom neighbor/boundary info into jacobi_kernel
#ifdef SOLUTION
        jacobi_kernel<dim_block_x, dim_block_y><<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
            a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm, top, iy_top_lower_boundary_idx, bottom, iy_bottom_upper_boundary_idx);
        CUDA_RT_CALL(hipGetLastError());
#else
        jacobi_kernel<dim_block_x, dim_block_y><<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
            a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));
#endif

        //TODO: add necessary inter PE synchronization
#ifdef SOLUTION
        nvshmemx_barrier_all_on_stream(compute_stream);
#endif

        if (calculate_norm) {
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost,
                                         compute_stream));
        }

        //TODO: Remove unnecessary MPI communication
#ifdef SOLUTION
#else
        // Apply periodic boundary conditions
        CUDA_RT_CALL(hipEventSynchronize(compute_done));
        PUSH_RANGE("MPI", 5)
        MPI_CALL(MPI_Sendrecv(a_new + iy_start * nx, nx, MPI_REAL_TYPE, top, 0,
                              a_new + (iy_end * nx), nx, MPI_REAL_TYPE, bottom, 0, MPI_COMM_WORLD,
                              MPI_STATUS_IGNORE));
        MPI_CALL(MPI_Sendrecv(a_new + (iy_end - 1) * nx, nx, MPI_REAL_TYPE, bottom, 0, a_new, nx,
                              MPI_REAL_TYPE, top, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));
        POP_RANGE
#endif

        if (calculate_norm) {
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            MPI_CALL(MPI_Allreduce(l2_norm_h, &l2_norm, 1, MPI_REAL_TYPE, MPI_SUM, MPI_COMM_WORLD));
            l2_norm = std::sqrt(l2_norm);

            if (!csv && 0 == rank && (iter % 100) == 0) {
                printf("%5d, %0.6f\n", iter, l2_norm);
            }
        }

        std::swap(a_new, a);
        iter++;
    }
    double stop = MPI_Wtime();
    POP_RANGE

    CUDA_RT_CALL(hipMemcpy(a_h + iy_start_global * nx, a + nx,
                            std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
                            hipMemcpyDeviceToHost));

    int result_correct = 1;
    for (int iy = iy_start_global; result_correct && (iy < iy_end_global); ++iy) {
        for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
            if (std::fabs(a_ref_h[iy * nx + ix] - a_h[iy * nx + ix]) > tol) {
                fprintf(stderr,
                        "ERROR on rank %d: a[%d * %d + %d] = %f does not match %f "
                        "(reference)\n",
                        rank, iy, nx, ix, a_h[iy * nx + ix], a_ref_h[iy * nx + ix]);
                result_correct = 0;
            }
        }
    }

    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));
    result_correct = global_result_correct;

    if (rank == 0 && result_correct) {
        if (csv) {
            printf("mpi, %d, %d, %d, %d, %d, 1, %f, %f\n", nx, ny, iter_max, nccheck, size,
                   (stop - start), runtime_serial);
        } else {
            printf("Num GPUs: %d.\n", size);
            printf(
                "%dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: %8.2f, "
                "efficiency: %8.2f \n",
                ny, nx, runtime_serial, size, (stop - start), runtime_serial / (stop - start),
                runtime_serial / (size * (stop - start)) * 100);
        }
    }
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    //TODO: Deallocated a_new and a from the NVSHMEM symmetric heap
#ifdef SOLUTION
    nvshmem_free(a_new);
    nvshmem_free(a);
#else
    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
#endif

    CUDA_RT_CALL(hipHostFree(a_h));
    CUDA_RT_CALL(hipHostFree(a_ref_h));

    //TODO: Finalize NVSHMEM
#ifdef SOLUTION
    nvshmem_finalize();
#endif
    MPI_CALL(MPI_Finalize());
    return (result_correct == 1) ? 0 : 1;
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel_single_gpu(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const bool calculate_norm) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;
        if (calculate_norm) {
            real residue = new_val - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print) {
    real* a;
    real* a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    real* l2_norm_d;
    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));

    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation: %d iterations on %d x %d mesh with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                  ((iy_end - iy_start) + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    real l2_norm = 1.0;
    bool calculate_norm;

    double start = MPI_Wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));

        calculate_norm = (iter % nccheck) == 0 || (iter % 100) == 0;
        jacobi_kernel_single_gpu<dim_block_x, dim_block_y><<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
            a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        if (calculate_norm) {
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost,
                                         compute_stream));
        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        if (calculate_norm) {
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            l2_norm = *l2_norm_h;
            l2_norm = std::sqrt(l2_norm);
            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = MPI_Wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}
