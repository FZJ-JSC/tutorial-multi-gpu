/** This example uses cudaManagedAllocation (available since CUDA 6.0) to allocate memory and manage
 *  the memory transfer to and from the device.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void scale(float alpha, float* a, float* c, int m){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < m){
        c[i] = alpha * a[i];
    }
}

int main(int argc, char** argv){

    int device = 0;
    if(argc > 1) {
        device = atoi(argv[1]);
        printf("Using device %d\n", device);
    }
    
    hipSetDevice(device);
    
    int m = 1000*2048;
    float alpha = 2.0;
    float tolerance = 1e-3f;
    float* a;
    float* c;

    hipMallocManaged(&a, m * sizeof(float));
    hipMallocManaged(&c, m * sizeof(float));
    for (int i = 0; i < m; ++i) a[i] = 1.0;

    dim3 blockDim(256);

    dim3 gridDim((m % 256) ? m / blockDim.x : m / blockDim.x + 1);

    // TODO: Augment code to tell runtime that data should be on GPU device before kernel launch
#ifdef SOLUTION
    cudaMemPrefetchAsync(c, m * sizeof(float), device);
    cudaMemPrefetchAsync(a, m * sizeof(float), device);
#else
#endif
    scale<<<gridDim, blockDim>>>(alpha, a, c, m);

    hipDeviceSynchronize();
   
    for (int i = 0; i < m; ++i){
        if (abs(c[i] - alpha * a[i]) > tolerance){
            printf("Failed! Element %d: c[%d] != %f a[%d] (%f != %f * %f)\n", i, i,alpha,i,c[i], alpha,a[i]);
            return 1;
        }
    }
    printf("Passed!\n");

    hipFree(a);
    hipFree(c);
    return 0;
}
